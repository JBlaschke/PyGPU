#include <cuda_hip_wrapper.h>
#include <pybind11/pybind11.h>


namespace py = pybind11;


template <class T> class ptr_wrapper {
    public:
        ptr_wrapper() : ptr(nullptr) {}
        ptr_wrapper(T * ptr) : ptr(ptr) {}
        ptr_wrapper(const ptr_wrapper& other) : ptr(other.ptr) {}
        T & operator* () const { return * ptr; }
        T * operator->() const { return   ptr; }
        T * get() const { return ptr; }
        void destroy() { delete ptr; }
        ~ptr_wrapper() { delete ptr; }
        T& operator[](std::size_t idx) const { return ptr[idx]; }
    private:
        T * ptr;
};


PYBIND11_MODULE(backend, m) {
    // TODO: this is a clumsy way to define data types -- clean this up a wee
    // bit in the future.
    py::class_<ptr_wrapper<int>>(m, "Int_t");

    m.def(
        "NewInt_t",
        []() {return ptr_wrapper<int>(new int); }
    );

    py::class_<ptr_wrapper<float>>(m, "Float_t");

    m.def(
        "NewFloat_t",
        []() {return ptr_wrapper<float>(new float); }
    );

    py::class_<ptr_wrapper<hipEvent_t>>(m, "hipEvent_t");

    m.def(
        "NewCudaEvent_t",
        []() {return ptr_wrapper<hipEvent_t>(new hipEvent_t); }
    );

    py::class_<ptr_wrapper<hipStream_t>>(m, "hipStream_t");

    m.def(
        "NewCudaStream_t",
        []() {return ptr_wrapper<hipStream_t>(new hipStream_t); }
    );

    py::class_<ptr_wrapper<int *>>(m, "IntPtr_t");

    m.def(
        "NewIntPtr_t",
        []() {return ptr_wrapper<int *>(new int *); }
    );

    py::class_<ptr_wrapper<double *>>(m, "DoublePtr_t");

    m.def(
        "NewDoublePtr_t",
        []() {return ptr_wrapper<double *>(new double *); }
    );




    m.def(
        "hipDeviceReset",
        []() {
            // TODO: use custom type for hipError_t
            return (int64_t) hipDeviceReset();
        }
    );


    m.def(
        "hipDeviceSynchronize",
        []() {
            // TODO: use custom type for hipError_t
            return (int64_t) hipDeviceSynchronize();
        }
    );


    m.def(
        "hipEventCreate",
        [](ptr_wrapper<hipEvent_t> event, unsigned int flags) {
            // TODO: use custom type for hipError_t
            return (int64_t) hipEventCreate(event.get(), flags);
        }
    );


    m.def(
        "hipEventElapsedTime",
        [](
            ptr_wrapper<float> ms,
            ptr_wrapper<hipEvent_t> start,
            ptr_wrapper<hipEvent_t> end
        ) {
            // TODO: use custom type for hipError_t
            return (int64_t) hipEventElapsedTime(ms.get(), * start, * end);
        }
    );


    m.def(
        "hipEventRecord",
        [](
            ptr_wrapper<hipEvent_t> event,
            ptr_wrapper<hipStream_t> end = 0
        ) {
            // TODO: use custom type for hipError_t
            return (int64_t) hipEventRecord(* event, * end);
        }
    );


    m.def(
        "hipEventSynchronize",
        [](ptr_wrapper<hipEvent_t> event) {
            // TODO: use custom type for hipError_t
            return (int64_t) hipEventSynchronize(* event);
        }
    );


    m.def(
        "hipFree",
        [](void * dev_ptr) {
            // TODO: use custom type for hipError_t
            return (int64_t) hipFree(dev_ptr);
        }
    );


    m.def(
        "hipHostFree",
        [](void * ptr) {
            // TODO: use custom type for hipError_t
            return (int64_t) hipHostFree(ptr);
        }
    );


    m.def(
        "hipGetDevice",
        [](ptr_wrapper<int> device) {
            // TODO: use custom type for hipError_t
            return (int64_t) hipGetDevice(device.get());
        }
    );


    m.def(
        "hipGetErrorName",
        [](ptr_wrapper<hipError_t> error) {
            return std::string(hipGetErrorName(* error));
        }
    );


    m.def(
        "hipGetErrorString",
        [](ptr_wrapper<hipError_t> error) {
            return std::string(hipGetErrorString(* error));
        }
    );


    m.def(
        "hipGetLastError",
        []() {
            // TODO: use custom type for hipError_t
            return (int64_t) hipGetLastError();
        }
    );


    // TODO: Template the argument data type
    m.def(
        "hipMalloc",
        [](ptr_wrapper<int *> dev_ptr, uint64_t size) {
            // TODO: use custom type for hipError_t
            return (int64_t) hipMalloc(dev_ptr.get(), size*sizeof(int));
        }
    );

    m.def(
        "hipMalloc",
        [](ptr_wrapper<double *> dev_ptr, uint64_t size) {
            // TODO: use custom type for hipError_t
            return (int64_t) hipMalloc(dev_ptr.get(), size*sizeof(double));
        }
    );


    // TODO: Template the argument data type
    m.def(
        "hipHostMalloc",
        [](ptr_wrapper<int *> dev_ptr, uint64_t size) {
            // TODO: use custom type for hipError_t
            return (int64_t) hipHostMalloc(dev_ptr.get(), size*sizeof(int));
        }
    );

    m.def(
        "hipHostMalloc",
        [](ptr_wrapper<double *> dev_ptr, uint64_t size) {
            // TODO: use custom type for hipError_t
            return (int64_t) hipHostMalloc(dev_ptr.get(), size*sizeof(double));
        }
    );

// //  __host__ ​hipError_t hipMemcpy ( void* dst, const void* src, size_t count, hipMemcpyKind kind ) 
// hipMemcpyDeviceToHost
// hipMemcpyHostToDevice
// 
//     // TODO: Template the argument data type to direct data
//     // using custom argument type
//     m.def(
//         "hipMemcpyDeviceToHost",
//         [](ptr_wrapper<int> dst, ptr_wrapper<int> src, uint64_t count) {
//             // TODO: use custom type for hipError_t
//             return (int64_t) hipMemcpy();
//         }
//     );
// 
//     m.def(
//         "hipHostMalloc",
//         [](ptr_wrapper<double *> dev_ptr, uint64_t size) {
//             // TODO: use custom type for hipError_t
//             return (int64_t) hipHostMalloc(dev_ptr.get(), size*sizeof(double));
//         }
//     );




    m.attr("major_version")   = py::int_(0);
    m.attr("minor_version")   = py::int_(1);
    m.attr("release_version") = py::int_(0);

    // Let the user know that this backend has been compiled _with_ CUDA support
    m.attr("cuda_enabled")            = py::bool_(true);
}
