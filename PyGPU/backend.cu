#include <data_type.h>
#include <device_wrapper.h>
#include <cuda_hip_wrapper.h>
#include <pybind11/pybind11.h>


namespace py = pybind11;


PYBIND11_MODULE(backend, m) {

    // Build all enumerations used internally by cuda bindings
    generate_enumeration(m);

    // Build all datatype wrapper bindings
    generate_datatype(m);

    py::class_<CudaError>(m, "hipError_t")
        .def(py::init<int>())
        .def("as_int", & CudaError::as_int)
        .def("__repr__",
            [](const CudaError & a) {
                return "<CudaError: 'code=" + std::to_string(a.as_int()) + "'>";
            }
        );

    // This needs to be defined so that the ptr_wrapper has something to return
    py::class_<ptr_wrapper<ihipEvent_t * >>(m, "_CUevent_st__ptr");

    py::class_<CudaEvent>(m, "hipEvent_t")
        .def(py::init<>())
        .def(py::init<int>())
        .def("get",
            [](CudaEvent & a) {
                return ptr_wrapper<hipEvent_t>(a.get());
            }
        )
        .def("last_status",
            [](const CudaEvent & a) {
                return CudaError(a.last_status());
            }
        );

    py::class_<CudaStream>(m, "hipStream_t")
        .def(py::init<>())
        .def(py::init<int>())
        .def("get",
            [](CudaStream & a) {
                return ptr_wrapper<hipStream_t>(a.get());
            }
        )
        .def("last_status",
            [](const CudaStream & a) {
                return CudaError(a.last_status());
            }
        );



    // TODO: this is a clumsy way to define data types -- clean this up a wee
    // bit in the future.

    py::class_<ptr_wrapper<int *>>(m, "IntPtr_t");

    m.def(
        "NewIntPtr_t",
        []() {return ptr_wrapper<int *>(new int *); }
    );

    py::class_<ptr_wrapper<double *>>(m, "DoublePtr_t");

    m.def(
        "NewDoublePtr_t",
        []() {return ptr_wrapper<double *>(new double *); }
    );


    m.def(
        "hipDeviceReset",
        []() {
            return CudaError(hipDeviceReset());
        }
    );


    m.def(
        "hipDeviceSynchronize",
        []() {
            return CudaError(hipDeviceSynchronize());
        }
    );


    m.def(
        "hipEventElapsedTime",
        [](CudaEvent & start, CudaEvent & end) {
            float ms;
            hipError_t err = hipEventElapsedTime(& ms, * start, * end);
            return std::make_tuple(ms, CudaError(err));
        }
    );


    m.def(
        "hipEventRecord",
        [](CudaEvent & event) {
            return CudaError(hipEventRecord(* event, 0));
        }
    );


    m.def(
        "hipEventRecord",
        [](CudaEvent & event, CudaStream & end) {
            return CudaError(hipEventRecord(* event, * end));
        }
    );


    m.def(
        "hipEventSynchronize",
        [](CudaEvent & event) {
            return CudaError(hipEventSynchronize(* event));
        }
    );


    m.def(
        "hipFree",
        [](void * dev_ptr) {
            return CudaError(hipFree(dev_ptr));
        }
    );


    m.def(
        "hipHostFree",
        [](void * ptr) {
            return CudaError(hipHostFree(ptr));
        }
    );


    m.def(
        "hipGetDevice",
        []() {
            int device;
            hipError_t err = hipGetDevice(& device);
            return std::make_tuple(device, CudaError(err));
        }
    );


    m.def(
        "hipGetErrorName",
        [](CudaError & error) {
            return std::string(hipGetErrorName(* error));
        }
    );


    m.def(
        "hipGetErrorString",
        [](CudaError & error) {
            return std::string(hipGetErrorString(* error));
        }
    );


    m.def(
        "hipGetLastError",
        []() {
            return CudaError(hipGetLastError());
        }
    );


    // TODO: Template the argument data type
    m.def(
        "hipMalloc",
        [](ptr_wrapper<int *> dev_ptr, uint64_t size) {
            return CudaError(hipMalloc(dev_ptr.get(), size*sizeof(int)));
        }
    );

    m.def(
        "hipMalloc",
        [](ptr_wrapper<double *> dev_ptr, uint64_t size) {
            return CudaError(hipMalloc(dev_ptr.get(), size*sizeof(double)));
        }
    );


    // TODO: Template the argument data type
    m.def(
        "hipHostMalloc",
        [](ptr_wrapper<int *> dev_ptr, uint64_t size) {
            return CudaError(hipHostMalloc(dev_ptr.get(), size*sizeof(int)));
        }
    );

    m.def(
        "hipHostMalloc",
        [](ptr_wrapper<double *> dev_ptr, uint64_t size) {
            // TODO: use custom type for hipError_t
            return CudaError(hipHostMalloc(dev_ptr.get(), size*sizeof(double)));
        }
    );

// //  __host__ ​hipError_t hipMemcpy ( void* dst, const void* src, size_t count, hipMemcpyKind kind ) 
// hipMemcpyDeviceToHost
// hipMemcpyHostToDevice
// 
//     // TODO: Template the argument data type to direct data
//     // using custom argument type
//     m.def(
//         "hipMemcpyDeviceToHost",
//         [](ptr_wrapper<int> dst, ptr_wrapper<int> src, uint64_t count) {
//             // TODO: use custom type for hipError_t
//             return (int64_t) hipMemcpy();
//         }
//     );
// 
//     m.def(
//         "hipHostMalloc",
//         [](ptr_wrapper<double *> dev_ptr, uint64_t size) {
//             // TODO: use custom type for hipError_t
//             return (int64_t) hipHostMalloc(dev_ptr.get(), size*sizeof(double));
//         }
//     );




    m.attr("major_version")   = py::int_(0);
    m.attr("minor_version")   = py::int_(1);
    m.attr("release_version") = py::int_(0);

    // Let the user know that this backend has been compiled _with_ CUDA support
    m.attr("cuda_enabled")            = py::bool_(true);
}
