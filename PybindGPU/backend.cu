#include <data_type.h>
#include <ptr_wrapper.h>
#include <cuda_hip_wrapper.h>
#include <pybind11/pybind11.h>

#include <error.h>
#include <event.h>
#include <stream.h>
#include <device_array.h>
#include <device_properties.h>


namespace py = pybind11;

PYBIND11_MODULE(backend, m) {

    // Build all enumerations used internally by cuda bindings
    generate_enumeration(m);
    // Build all datatype wrapper bindings
    generate_datatype(m);

    generate_device_array(m);

    generate_cuda_error(m);
    generate_cuda_event(m);
    generate_cuda_stream(m);

    m.def(
        "hipDeviceReset",
        []() {
            return CudaError(hipDeviceReset());
        }
    );

    m.def(
        "hipDeviceSynchronize",
        []() {
            return CudaError(hipDeviceSynchronize());
        }
    );

    m.def(
        "hipEventElapsedTime",
        [](CudaEvent & start, CudaEvent & end) {
            float ms;
            hipError_t err = hipEventElapsedTime(& ms, * start, * end);
            return std::make_tuple(ms, CudaError(err));
        }
    );

    m.def(
        "hipEventRecord",
        [](CudaEvent & event) {
            return CudaError(hipEventRecord(* event, 0));
        }
    );

    m.def(
        "hipEventRecord",
        [](CudaEvent & event, CudaStream & end) {
            return CudaError(hipEventRecord(* event, * end));
        }
    );

    m.def(
        "hipEventSynchronize",
        [](CudaEvent & event) {
            return CudaError(hipEventSynchronize(* event));
        }
    );

    m.def(
        "hipGetDevice",
        []() {
            int device;
            hipError_t err = hipGetDevice(& device);
            return std::make_tuple(device, CudaError(err));
        }
    );

    m.def(
        "hipSetDevice",
        [](int device) {
            return CudaError(hipSetDevice(device));
        }
    );

    m.def(
        "hipGetErrorName",
        [](CudaError & error) {
            return std::string(hipGetErrorName(* error));
        }
    );

    m.def(
        "hipGetErrorString",
        [](CudaError & error) {
            return std::string(hipGetErrorString(* error));
        }
    );

    m.def(
        "hipGetLastError",
        []() {
            return CudaError(hipGetLastError());
        }
    );

    // This needs to be defined so that the ptr_wrapper has something to return
    py::class_<ptr_wrapper<hipDeviceProp_t>>(m, "_CudaDeviceProp__ptr");

    py::class_<DeviceProperties>(m, "hipDeviceProp_t")
        .def(py::init<int>())
        .def("get",
            [](DeviceProperties & a) {
                return ptr_wrapper<hipDeviceProp_t>(a.get());
            }
        )
        .def("name",
            [](DeviceProperties & a) {
                std::string s(a.get()->name);
                return s;
            }
        )
#ifndef USE_HIP
        .def("uuid",
            [](DeviceProperties & a) {
                std::string s = mem_to_string(
                    reinterpret_cast<void *>(& a.get()->uuid), 16
                );
                return s;
            }
        )
#endif
        .def("pciBusID",
            [](DeviceProperties & a) {
                return a.get()->pciBusID;
            }
        )
        .def("pciDeviceID",
            [](DeviceProperties & a) {
                return a.get()->pciBusID;
            }
        )
        .def("pciDomainID",
            [](DeviceProperties & a) {
                return a.get()->pciBusID;
            }
        )
        .def("last_status",
            [](const DeviceProperties & a) {
                return CudaError(a.last_status());
            }
        );

        m.def(
            "hipGetDeviceCount",
            []() {
                int device;
                hipError_t err = hipGetDeviceCount(& device);
                return std::make_tuple(device, CudaError(err));
            }
        );

    m.attr("major_version")   = py::int_(0);
    m.attr("minor_version")   = py::int_(2);
    m.attr("release_version") = py::int_(1);

    // Let the user know if PybindGPU has been built in HIP mode instead of CUDA
#ifdef USE_HIP
    m.attr("use_hip") = py::bool_(true);
#else
    m.attr("use_hip") = py::bool_(false);
#endif

    // Let the user know that this backend has been compiled _with_ CUDA support
    m.attr("cuda_enabled") = py::bool_(true);
}
