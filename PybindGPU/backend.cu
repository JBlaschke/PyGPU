#include <data_type.h>
#include <ptr_wrapper.h>
#include <device_wrapper.h>
#include <device_properties.h>
#include <cuda_hip_wrapper.h>
#include <pybind11/pybind11.h>


namespace py = pybind11;


PYBIND11_MODULE(backend, m) {

    // Build all enumerations used internally by cuda bindings
    generate_enumeration(m);

    // Build all datatype wrapper bindings
    generate_datatype(m);

    py::class_<CudaError>(m, "hipError_t")
        .def(py::init<int>())
        .def("as_int", & CudaError::as_int)
        .def("__repr__",
            [](const CudaError & a) {
                return "<CudaError: 'code=" + std::to_string(a.as_int()) + "'>";
            }
        );

    // This needs to be defined so that the ptr_wrapper has something to return
    py::class_<ptr_wrapper<hipEvent_t>>(m, "_CUevent_st__ptr");

    py::class_<CudaEvent>(m, "hipEvent_t")
        .def(py::init<>())
        .def(py::init<int>())
        .def("get",
            [](CudaEvent & a) {
                return ptr_wrapper<hipEvent_t>(a.get());
            }
        )
        .def("last_status",
            [](const CudaEvent & a) {
                return CudaError(a.last_status());
            }
        );

    py::class_<CudaStream>(m, "hipStream_t")
        .def(py::init<>())
        .def(py::init<int>())
        .def("get",
            [](CudaStream & a) {
                return ptr_wrapper<hipStream_t>(a.get());
            }
        )
        .def("last_status",
            [](const CudaStream & a) {
                return CudaError(a.last_status());
            }
        );

    m.def(
        "hipDeviceReset",
        []() {
            return CudaError(hipDeviceReset());
        }
    );

    m.def(
        "hipDeviceSynchronize",
        []() {
            return CudaError(hipDeviceSynchronize());
        }
    );

    m.def(
        "hipEventElapsedTime",
        [](CudaEvent & start, CudaEvent & end) {
            float ms;
            hipError_t err = hipEventElapsedTime(& ms, * start, * end);
            return std::make_tuple(ms, CudaError(err));
        }
    );

    m.def(
        "hipEventRecord",
        [](CudaEvent & event) {
            return CudaError(hipEventRecord(* event, 0));
        }
    );

    m.def(
        "hipEventRecord",
        [](CudaEvent & event, CudaStream & end) {
            return CudaError(hipEventRecord(* event, * end));
        }
    );

    m.def(
        "hipEventSynchronize",
        [](CudaEvent & event) {
            return CudaError(hipEventSynchronize(* event));
        }
    );

    m.def(
        "hipGetDevice",
        []() {
            int device;
            hipError_t err = hipGetDevice(& device);
            return std::make_tuple(device, CudaError(err));
        }
    );

    m.def(
        "hipSetDevice",
        [](int device) {
            return CudaError(hipSetDevice(device));
        }
    );

    m.def(
        "hipGetErrorName",
        [](CudaError & error) {
            return std::string(hipGetErrorName(* error));
        }
    );

    m.def(
        "hipGetErrorString",
        [](CudaError & error) {
            return std::string(hipGetErrorString(* error));
        }
    );

    m.def(
        "hipGetLastError",
        []() {
            return CudaError(hipGetLastError());
        }
    );

    // This needs to be defined so that the ptr_wrapper has something to return
    py::class_<ptr_wrapper<hipDeviceProp_t>>(m, "_CudaDeviceProp__ptr");

    py::class_<DeviceProperties>(m, "hipDeviceProp_t")
        .def(py::init<int>())
        .def("get",
            [](DeviceProperties & a) {
                return ptr_wrapper<hipDeviceProp_t>(a.get());
            }
        )
        .def("name",
            [](DeviceProperties & a) {
                std::string s(a.get()->name);
                return s;
            }
        )
#ifndef USE_HIP
        .def("uuid",
            [](DeviceProperties & a) {
                std::string s = mem_to_string(
                    reinterpret_cast<void *>(& a.get()->uuid), 16
                );
                return s;
            }
        )
#endif
        .def("pciBusID",
            [](DeviceProperties & a) {
                return a.get()->pciBusID;
            }
        )
        .def("pciDeviceID",
            [](DeviceProperties & a) {
                return a.get()->pciBusID;
            }
        )
        .def("pciDomainID",
            [](DeviceProperties & a) {
                return a.get()->pciBusID;
            }
        )
        .def("last_status",
            [](const DeviceProperties & a) {
                return CudaError(a.last_status());
            }
        );

        m.def(
            "hipGetDeviceCount",
            []() {
                int device;
                hipError_t err = hipGetDeviceCount(& device);
                return std::make_tuple(device, CudaError(err));
            }
        );

    m.attr("major_version")   = py::int_(0);
    m.attr("minor_version")   = py::int_(2);
    m.attr("release_version") = py::int_(1);

    // Let the user know if PybindGPU has been built in HIP mode instead of CUDA
#ifdef USE_HIP
    m.attr("use_hip") = py::bool_(true);
#else
    m.attr("use_hip") = py::bool_(false);
#endif

    // Let the user know that this backend has been compiled _with_ CUDA support
    m.attr("cuda_enabled") = py::bool_(true);
}
